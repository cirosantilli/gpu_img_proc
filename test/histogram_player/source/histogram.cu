#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <histogram.h>
#include <cuda_gl_interop.h>

texture<uint8_t, 2, hipReadModeElementType> tex_r;
texture<uchar4, 2, hipReadModeElementType> tex_rgba;

__global__ void comp_histogram(GLint* hist)
{
    uint8_t red;
    int i, x, y;
    __shared__ int temp_hist[256];

    i = (blockDim.x*threadIdx.y)+threadIdx.x;

    temp_hist[i] = 0;

    __syncthreads();
  
    x = (blockIdx.x*blockDim.x+threadIdx.x);
    y = (blockIdx.y*blockDim.y+threadIdx.y);

    red = tex2D(tex_r, x, y);

    atomicAdd(&temp_hist[red], 1);

    __syncthreads();

    atomicAdd(&hist[i], temp_hist[i]);
}

__global__ void comp_histogram(GLint* hist_r, GLint* hist_g, GLint* hist_b)
{
    int i, x, y;
    uint8_t red, green, blue;

    __shared__ int temp_hist_r[256];
    __shared__ int temp_hist_g[256];
    __shared__ int temp_hist_b[256];

    i = (blockDim.x*threadIdx.y)+threadIdx.x;

    temp_hist_r[i] = 0;
    temp_hist_g[i] = 0;
    temp_hist_b[i] = 0;

    __syncthreads();
  
    x = (blockIdx.x*blockDim.x+threadIdx.x);
    y = (blockIdx.y*blockDim.y+threadIdx.y);

    red = tex2D(tex_rgba, x, y).x;
    green = tex2D(tex_rgba, x, y).y;
    blue = tex2D(tex_rgba, x, y).z;

    atomicAdd(&temp_hist_r[red], 1);
    atomicAdd(&temp_hist_g[green], 1);
    atomicAdd(&temp_hist_b[blue], 1);

    __syncthreads();

    atomicAdd(&hist_r[i], temp_hist_r[i]);
    atomicAdd(&hist_g[i], temp_hist_g[i]);
    atomicAdd(&hist_b[i], temp_hist_b[i]);
}

__global__ void max_value(GLint* hist, GLint* max, GLint count)
{
    int p, q;
    int i, j, k, l;
    __shared__ int temp[2][256];
    
    temp[0][threadIdx.x] = hist[threadIdx.x];

    __syncthreads();

    i = 0;
    j = 1;
    l = count/2;
    while (l > 0)
    {
        if (threadIdx.x < l)
        {
            p = (2*threadIdx.x); q = p+1;
            temp[j][threadIdx.x] = (temp[i][p] < temp[i][q])?temp[i][q]:temp[i][p];
        }
        l /= 2;
        k = i;
        i = j;
        j = k;
        __syncthreads();
    }
    *max = temp[i][0];
}

__global__ void convert_to_coords(GLint* hist, GLfloat* coord, GLint* max)
{
    int p, q;
    p = (2*threadIdx.x); q = p+1;
    coord[p] = ((GLfloat)threadIdx.x)/256.0f;
    coord[q] = ((GLfloat)hist[threadIdx.x])/((GLfloat)*max);
}


void compute_histogram(unsigned int texture, unsigned int hist_obj, int width, int height)
{
    hipError_t err;
    GLfloat* dev_hist = 0;

    hipArray* array;
    hipGraphicsResource* res;

    cudaGLRegisterBufferObject(hist_obj);
    cudaGLMapBufferObject((void **)&dev_hist, hist_obj);

    err = hipGraphicsGLRegisterImage(&res, texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly);
    if (err != hipSuccess)
    {
        printf("hipGraphicsGLRegisterImage Failed: %s", hipGetErrorString(hipGetLastError()));
        exit(0);
    }

    hipGraphicsMapResources(1, &res);

    hipChannelFormatDesc chan_desc = hipCreateChannelDesc<uint8_t>();
    err = hipGraphicsSubResourceGetMappedArray(&array, res, 0, 0);
    if (err != hipSuccess)
    {
        printf("hipGraphicsSubResourceGetMappedArray Failed: %s", hipGetErrorString(hipGetLastError()));
        exit(0);
    }

    if (hipBindTextureToArray(&tex_r, array, &chan_desc) != hipSuccess) {
	    printf("Failed to bind texture - %s\n", hipGetErrorString(hipGetLastError()));
	    exit(0);
    }

    GLint* dev_hist_mem = 0;
    hipMalloc(&dev_hist_mem, (1+256)*sizeof(GLint));
    hipMemset(dev_hist_mem, 0, (1+256)*sizeof(GLint)); 

    dim3 bsize(16, 16);
    dim3 gsize(width/bsize.x, height/bsize.y);
    comp_histogram<<<gsize, bsize>>>(dev_hist_mem);
    max_value<<<1, 256>>>(dev_hist_mem, &dev_hist_mem[256], 256);
    convert_to_coords<<<1, 256>>>(dev_hist_mem, dev_hist, &dev_hist_mem[256]);
    hipDeviceSynchronize();

    hipFree(dev_hist_mem);

    hipUnbindTexture(tex_r);
    cudaGLUnmapBufferObject(hist_obj);
    cudaGLUnregisterBufferObject(hist_obj);
    
    hipGraphicsUnmapResources(1, &res);
    hipGraphicsUnregisterResource(res);
}

void compute_histogram(unsigned int texture, unsigned int* hist_obj, int width, int height)
{
    hipError_t err;
    GLfloat* dev_hist[3] = {0, 0, 0};

    hipArray* array;
    hipGraphicsResource* res;

    for (int i = 0; i < 3; i++)
    {
        cudaGLRegisterBufferObject(hist_obj[i]);
        cudaGLMapBufferObject((void **)&dev_hist[i], hist_obj[i]);
    }

    err = hipGraphicsGLRegisterImage(&res, texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly);
    if (err != hipSuccess)
    {
        printf("hipGraphicsGLRegisterImage Failed: %s", hipGetErrorString(hipGetLastError()));
        exit(0);
    }

    hipGraphicsMapResources(1, &res);

    hipChannelFormatDesc chan_desc = hipCreateChannelDesc<uchar4>();
    err = hipGraphicsSubResourceGetMappedArray(&array, res, 0, 0);
    if (err != hipSuccess)
    {
        printf("hipGraphicsSubResourceGetMappedArray Failed: %s", hipGetErrorString(hipGetLastError()));
        exit(0);
    }

    if (hipBindTextureToArray(&tex_rgba, array, &chan_desc) != hipSuccess) {
	    printf("Failed to bind texture - %s\n", hipGetErrorString(hipGetLastError()));
	    exit(0);
    }

    GLint* dev_hist_mem = 0;
    hipMalloc(&dev_hist_mem, (3+(256*3))*sizeof(GLint));
    hipMemset(dev_hist_mem, 0, (3+(256*3))*sizeof(GLint));

    hipStream_t stream1, stream2, stream3;
    hipStreamCreate(&stream1); 
    hipStreamCreate(&stream2); 
    hipStreamCreate(&stream3); 

    dim3 bsize(16, 16);
    dim3 gsize(width/bsize.x, height/bsize.y);
    comp_histogram<<<gsize, bsize>>>(dev_hist_mem, &dev_hist_mem[256], &dev_hist_mem[512]);
    max_value<<<1, 256, 0, stream1>>>(dev_hist_mem, &dev_hist_mem[768], 256);
    max_value<<<1, 256, 0, stream2>>>(&dev_hist_mem[256], &dev_hist_mem[769], 256);
    max_value<<<1, 256, 0, stream3>>>(&dev_hist_mem[512], &dev_hist_mem[770], 256);
    convert_to_coords<<<1, 256, 0, stream1>>>(dev_hist_mem, dev_hist[0], &dev_hist_mem[768]);
    convert_to_coords<<<1, 256, 0, stream2>>>(&dev_hist_mem[256], dev_hist[1], &dev_hist_mem[769]);
    convert_to_coords<<<1, 256, 0, stream3>>>(&dev_hist_mem[512], dev_hist[2], &dev_hist_mem[770]);
    //hipDeviceSynchronize();

    hipFree(dev_hist_mem);
    hipStreamDestroy(stream1); 
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);

    hipUnbindTexture(tex_rgba);
    for (int i = 0; i < 3; i++)
    {
        cudaGLUnmapBufferObject(hist_obj[i]);
        cudaGLUnregisterBufferObject(hist_obj[i]);
    }

    hipGraphicsUnmapResources(1, &res);
    hipGraphicsUnregisterResource(res);
}

void print_cuda_device_info()
{
    int count = 0;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&count);

    printf("\nCUDA Device Count: %d", count); 
    for (int i = 0; i < count; i++)
    {
        hipGetDeviceProperties(&prop, i);
        printf("\nDevice: %d", i);
        printf("\nName: %s", prop.name);
        printf("\nRevision: Major: %d, Minor: %d", prop.major, prop.minor);
        printf("\nWarp Size: %d", prop.warpSize);
        printf("\nMemory Bus width: %d", prop.memoryBusWidth);
        printf("\nMemory Clock Rate: %d", prop.memoryClockRate);
        printf("\nConcurrent Kernels: %d", prop.concurrentKernels);
        printf("\nMultiprocessor count: %d", prop.multiProcessorCount);
        printf("\nTotal Global Memory: %d", (int)prop.totalGlobalMem);
        printf("\nTotal Constant Memory: %d", (int)prop.totalConstMem);
        printf("\nShared Memory per Block: %d", (int)prop.sharedMemPerBlock);
        printf("\nMax grid dimensions: (%d, %d, %d)", prop.maxGridSize[0], 
                                                       prop.maxGridSize[1], 
                                                       prop.maxGridSize[2]);
        printf("\nMax threads per block: %d", prop.maxThreadsPerBlock);
        printf("\nMax threads dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], 
                                                       prop.maxThreadsDim[1], 
                                                       prop.maxThreadsDim[2]);
    }
}

